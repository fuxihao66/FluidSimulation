#include "hip/hip_runtime.h"

struct TriangleMesh{
    
    #ifdef WRITE
    RWBuffer<float3> Vertices;
    #else
    Buffer<float3>   Vertices;
    #endif

    #ifdef WRITE
    RWBuffer<uint>   Indices;
    #else
    Buffer<uint>     Indices;
    #endif
    
    uint TriangleNums;
    uint VertexNums;

    #ifdef WRITE
    void AppendVertexAtomic(){
        Vertices[];
    }
    void AppendTriangleAtomic(){
        
    }
    #endif
};

struct HashTable{
    
};

struct KeyValueMap{

};