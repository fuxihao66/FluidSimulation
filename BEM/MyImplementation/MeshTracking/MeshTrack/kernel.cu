#include "hip/hip_runtime.h"

#define TINYOBJLOADER_IMPLEMENTATION
#include "tiny_obj_loader.h"
#include <vector>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

struct Vertex {
    float3 Position;
    float3 Velocity;
};
struct IntersectionHashMap {
    float* cudaMemoryBuffer;
    unsigned int entriesPerCell;
    IntersectionHashMap(float far, float near, float left, float right, float bottom, float up, float gridSize, unsigned int entries) {
        //IntersectionHashMap(unsigned int xNum, unsigned int yNum, unsigned int zNum, unsigned int entries) {
        unsigned int xNum = ceil((near - far) / gridSize);
        unsigned int yNum = ceil((right - left) / gridSize);
        unsigned int zNum = ceil((up - bottom) / gridSize);
        entriesPerCell = entries;
        hipError_t cudaStatus = hipMalloc((void**)&cudaMemoryBuffer, (xNum * yNum * zNum) * (entries + 1) * sizeof(unsigned int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
        }
    }
};

__global__ void AdvectMeshKernel(float* meshVertices, const float timeStep, const int vertexNumber) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    //c[i] = a[i] + b[i];
    if (i < vertexNumber) {
        meshVertices[6 * i + 0] += timeStep * meshVertices[6 * i + 3];
        meshVertices[6 * i + 1] += timeStep * meshVertices[6 * i + 4];
        meshVertices[6 * i + 2] += timeStep * meshVertices[6 * i + 5];
    }
    
}

__global__ void IntersectionTestKernel1(float* meshVertices, unsigned int* meshIndices, unsigned int triangleNum, IntersectionHashMap* hashMap){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < triangleNum) {
        unsigned int vertexIndex0 = meshIndices[3 * i + 0];
        unsigned int vertexIndex1 = meshIndices[3 * i + 1];
        unsigned int vertexIndex2 = meshIndices[3 * i + 2];

        // ��0�������xyz
        meshVertices[6 * vertexIndex0 + 0];
        meshVertices[6 * vertexIndex0 + 1];
        meshVertices[6 * vertexIndex0 + 2];

        // ��1�������xyz
        meshVertices[6 * vertexIndex1 + 0];
        meshVertices[6 * vertexIndex1 + 1];
        meshVertices[6 * vertexIndex1 + 2];

        // ��2�������xyz
        meshVertices[6 * vertexIndex2 + 0];
        meshVertices[6 * vertexIndex2 + 1];
        meshVertices[6 * vertexIndex2 + 2];

        // �����������������ε�����cell
        // ÿ��cell��entry����������������
        for each cell{
            hashMap
        }
    }
}

__device__ bool TriangleIntersection(unsigned int tri0Index0, unsigned int tri0Index1, unsigned int tri0Index2, unsigned int tri1Index0, unsigned int tri1Index1, unsigned int tri1Index2, float* meshVertices) {

    return true;
}

__global__ void IntersectionTestKernel2(float* meshVertices, unsigned int* meshIndices, unsigned int* deleteBuffer, unsigned int triangleNum){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < triangleNum) {
        unsigned int vertexIndex0 = meshIndices[3 * i + 0];
        unsigned int vertexIndex1 = meshIndices[3 * i + 1];
        unsigned int vertexIndex2 = meshIndices[3 * i + 2];

        // ��0�������xyz
        meshVertices[6 * vertexIndex0 + 0];
        meshVertices[6 * vertexIndex0 + 1];
        meshVertices[6 * vertexIndex0 + 2];

        // ��1�������xyz
        meshVertices[6 * vertexIndex1 + 0];
        meshVertices[6 * vertexIndex1 + 1];
        meshVertices[6 * vertexIndex1 + 2];

        // ��2�������xyz
        meshVertices[6 * vertexIndex2 + 0];
        meshVertices[6 * vertexIndex2 + 1];
        meshVertices[6 * vertexIndex2 + 2];

        // �����������������ε�����cell
        // ÿ��cell��entry�������������������������ν����ཻ����
        
        for (possible grid) {
            if grid intersect{
                for (triIndex in grid) {
                    unsigned int triVertexIndex0 = meshIndices[3 * triIndex];
                    unsigned int triVertexIndex1 = meshIndices[3 * triIndex + 1];
                    unsigned int triVertexIndex2 = meshIndices[3 * triIndex + 2];

                    // ���ݶ����ж�


                    // �����ཻ�ж�
                    bool hasIntersection = TriangleIntersection(vertexIndex0, vertexIndex1, vertexIndex2, triVertexIndex0, triVertexIndex1, triVertexIndex2, meshVertices);
                    if (hasIntersection) {
                        atomicExch(deleteBuffer[triIndex], 1);
                        atomicExch(deleteBuffer[i], 1);
                    }
                }
            }
        }
    }
}
/*
__global__ void InsideVolumeKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void DeleteTriangleAndGenBoundaryKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
__device__ void ManifoldEnforceThreadFunction() {

}

__global__ void ManifoldEnforcementKernel(int* c, const int* a, const int* b){
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

    ManifoldEnforceThreadFunction << < >> > ();   // kernel�е���kernel
}


__global__ void IdentifyHoleKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

}
__global__ void PairHoleKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

}
__global__ void FillHoleKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

}
__global__ void DeleteTriangleAndUpdateBoundaryKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

}


__global__ void ImproveMeshKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

}

__global__ void RemoveTrianglesKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

}
__global__ void RemoveVerticesKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

}
*/




hipError_t MeshTrack(float* cudaVertexBuffer, unsigned int* cudaIndexBuffer, unsigned int* deleteBuffer, unsigned int vertexNum, unsigned int triangleNum, IntersectionHashMap* hashMap) {
    // 1. detect intersection
    {
        dim3 ThreadPerBlock(512, 1, 1);
        dim3 NumBlock(ceil(triangleNum/float(ThreadPerBlock.x)), 1, 1);
        IntersectionTestKernel1<<<NumBlock, ThreadPerBlock>>>(cudaVertexBuffer, cudaIndexBuffer, deleteBuffer, triangleNum, hashMap);
        IntersectionTestKernel2<<<NumBlock, ThreadPerBlock>>>();
    }

    

    /*
    // 2. detect inside volume
    InsideVolumeKernel << < >> > ();

    // 3. delete and generate boundary list
    DeleteTriangleAndGenBoundaryKernel << < >> > ();

    // 4. loop
    while (1) {
        ManifoldEnforcementKernel << < >> > ();


        if () {
            break;
        }


    }


    ImproveMeshKernel << < >> > ();
    RemoveTrianglesKernel << < >> > ();
    RemoveVerticesKernel << < >> > ();
    */
}

int main()
{

    /*const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };*/
    const unsigned int MaxFrameCount = 10;
    const float timeStep = 0.01f;

    const float leftBound = -1.5f;  // y
    const float rightBound = 1.5f;
    const float bottomBound = -1.5f; //z
    const float upBound = 1.5f;
    const float farBound = -1.5f; //x
    const float nearBound = 1.5f;

    const float gridSize = 0.01f;   // ��Ҫ����ߵ�3��
    const unsigned int entrySize = 100; // ÿ��cell������ٸ�������




    // ����ˮ��
    std::vector<Vertex> vertices;
    std::vector<unsigned int> indices;
    unsigned int vertexNum = 0;
    unsigned int indexNum = 0;

    {
        tinyobj::attrib_t attrib;
        std::vector<tinyobj::shape_t> shapes;
        std::vector<tinyobj::material_t> materials;
        std::string warn;
        std::string err;
        std::string filePath = "data/wooden_sphere.obj";
        bool ret = tinyobj::LoadObj(&attrib, &shapes, &materials, &warn, &err, filePath.c_str(),
            NULL, true);

        vertexNum = attrib.vertices.size() / 3 * 2;
        vertices.resize(vertexNum);
        for (size_t v = 0; v < attrib.vertices.size() / 3; v++) {
            vertices[v].Position.x = attrib.vertices[3 * v + 0];
            vertices[v].Position.y = attrib.vertices[3 * v + 1] - 0.52f;
            vertices[v].Position.z = attrib.vertices[3 * v + 2];
            vertices[v].Velocity.x = 0.0f;
            vertices[v].Velocity.y = 0.5f;
            vertices[v].Velocity.z = 0.0f;
        }

        for (size_t v = attrib.vertices.size() / 3; v < vertexNum; v++) {
            vertices[v].Position.x = attrib.vertices[3 * (v - attrib.vertices.size() / 3) + 0];
            vertices[v].Position.y = attrib.vertices[3 * (v - attrib.vertices.size() / 3) + 1] + 0.52f;
            vertices[v].Position.z = attrib.vertices[3 * (v - attrib.vertices.size() / 3) + 2];
            vertices[v].Velocity.x = 0.0f;
            vertices[v].Velocity.y = -0.5f;
            vertices[v].Velocity.z = 0.0f;
        }


        indices.resize(shapes[0].mesh.num_face_vertices.size() * 3 * 3);
        for (size_t i = 0; i < shapes.size(); i++) {

            size_t index_offset = 0;

            // For each face
            for (size_t f = 0; f < shapes[i].mesh.num_face_vertices.size(); f++) {
                size_t fnum = shapes[i].mesh.num_face_vertices[f];

                // For each vertex in the face
                for (size_t v = 0; v < fnum; v++) {
                    tinyobj::index_t idx = shapes[i].mesh.indices[index_offset + v];
                    indices[index_offset + v] = idx.vertex_index;
                    indexNum += 1;
                }

                index_offset += fnum;
            }
        }

        for (size_t i = 0; i < indexNum; i++) {
            indices[indexNum + i] = indices[i] + attrib.vertices.size() / 3;
        }
        indexNum *= 2;
    }

    hipError_t cudaStatus = hipSetDevice(0);
    float* cudaVertexBuffer;
    unsigned int* cudaIndexBuffer;

    {//����mesh��Դ    
    // TODO: ����������Ŀռ�  ��������
        cudaStatus = hipMalloc((void**)&cudaVertexBuffer, 3 * vertexNum * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
        }
        cudaStatus = hipMalloc((void**)&cudaIndexBuffer, 3 * indexNum * sizeof(unsigned int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
        }

        cudaStatus = hipMemcpy(cudaVertexBuffer, (void*)vertices.data(), vertexNum * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
        }

        cudaStatus = hipMemcpy(cudaIndexBuffer, (void*)indices.data(), indexNum * sizeof(unsigned int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
        }
    }
    
    unsigned int* deleteBuffer;
    
    {// �����м�����ռ�
        cudaStatus = hipMalloc((void**)&deleteBuffer, indexNum * sizeof(unsigned int));  // ����Ŀռ������������ε��������Է���������������
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
        }
    }
    IntersectionHashMap* spatialMap = new IntersectionHashMap(farBound, nearBound, leftBound, rightBound, bottomBound, upBound, gridSize, entrySize);


    // ��ʼ����
    for (int i = 0; i < MaxFrameCount; i++) {
        // advect mesh 
        dim3 NumBlock;
        dim3 ThreadPerBlock;
        ThreadPerBlock.x = 64;
        ThreadPerBlock.y = 1;
        ThreadPerBlock.z = 1;
        NumBlock.x = ceil(vertexNum / float(ThreadPerBlock.x));
        NumBlock.y = 1;
        NumBlock.z = 1;
        AdvectMeshKernel<<<NumBlock, ThreadPerBlock >>>(cudaVertexBuffer, timeStep, vertexNum);
        /*cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        }*/
        


        // begin mesh operate

        cudaStatus = MeshTrack(cudaVertexBuffer, cudaIndexBuffer, deleteBuffer, vertexNum, indexNum / 3, spatialMap);


        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            return 1;
        }

        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return 1;
        }
    }


   

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
